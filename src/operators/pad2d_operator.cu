#include "hip/hip_runtime.h"
///@file pad2d_operator.cu
///@brief Operator that pads an image given with symmetric boundary conndition
///@author Erich Kobler <erich.kobler@icg.tugraz.at>
///@date 01.202


#include "utils.h"
#include "tensor/d_tensor.h"
#include "pad2d_operator.h"

#include "reduce.cuh"


inline __device__ int symPixel(int x, int width)
{
  int x_ = x;
  if (x < 0)
    x_ = abs(x);
  else if (x >= width)
    x_ = 2 * width - x - 2;
  return x_;
}


template <typename T>
__global__ void pad2d(
    typename optox::DTensor<T, 3>::Ref out,
    const typename optox::DTensor<T, 3>::ConstRef in,
    int left, int top)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x < out.size_[2] && y < out.size_[1] && z < out.size_[0])
    {
        // compute the corresponding index 
        const int x_in = symPixel(x - left, in.size_[2]);
        const int y_in = symPixel(y - top, in.size_[1]);
        out(z, y, x) = in(z, y_in, x_in);
    }
}


template<typename T>
void optox::Pad2dOperator<T>::computeForward(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto x = this->template getInput<T, 3>(0, inputs);
    auto out = this->template getOutput<T, 3>(0, outputs);

    if (x->size()[0] != out->size()[0] || 
        x->size()[1]+this->top_+this->bottom_ != out->size()[1]||
        x->size()[2]+this->left_+this->right_ != out->size()[2])
        THROW_OPTOXEXCEPTION("Pad2dOperator: input and output size do not match!");

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid = dim3(divUp(out->size()[2], dim_block.x),
                         divUp(out->size()[1], dim_block.y),
                         divUp(out->size()[0], dim_block.z));

    pad2d<T> <<<dim_grid, dim_block, 0, this->stream_>>>(*out, *x, 
        this->left_, this->top_);
    OPTOX_CUDA_CHECK;
}


template <typename T>
__global__ void pad2d_grad(
    typename optox::DTensor<T, 3>::Ref grad_in,
    const typename optox::DTensor<T, 3>::ConstRef grad_out,
    int left, int top)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x < grad_out.size_[2] && y < grad_out.size_[1] && z < grad_out.size_[0])
    {
        // compute the corresponding index 
        const int x_in = symPixel(x - left, grad_in.size_[2]);
        const int y_in = symPixel(y - top, grad_in.size_[1]);
        atomicAdd(&grad_in(z, y_in, x_in), grad_out(z, y, x));
    }
}


template<typename T>
void optox::Pad2dOperator<T>::computeAdjoint(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto grad_out = this->template getInput<T, 3>(0, inputs);

    auto grad_x = this->template getOutput<T, 3>(0, outputs);

    // clear the weights gradient
    grad_x->fill(0);

    if (grad_x->size()[0] != grad_out->size()[0] || 
        grad_x->size()[1]+this->top_+this->bottom_ != grad_out->size()[1]||
        grad_x->size()[2]+this->left_+this->right_ != grad_out->size()[2])
        THROW_OPTOXEXCEPTION("Pad2dOperator-adjoint: input and output size do not match!");

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid = dim3(divUp(grad_out->size()[2], dim_block.x),
                         divUp(grad_out->size()[1], dim_block.y),
                         divUp(grad_out->size()[0], dim_block.z));

    pad2d_grad<T> <<<dim_grid, dim_block, 0, this->stream_>>>(*grad_x, *grad_out, 
        this->left_, this->top_);
    OPTOX_CUDA_CHECK;
}

#define REGISTER_OP(T) \
    template class optox::Pad2dOperator<T>;

OPTOX_CALL_REAL_NUMBER_TYPES(REGISTER_OP);
#undef REGISTER_OP
