#include "hip/hip_runtime.h"
///@file nabla_operator.cu
///@brief Operator that computes the forward differences along all dimensions
///@author Erich Kobler <erich.kobler@icg.tugraz.at>
///@date 09.07.2018


#include "utils.h"
#include "tensor/d_tensor.h"
#include "nabla_operator.h"

template<typename T>
__global__ void forward_differences(
    typename optox::DTensor<T, 3>::Ref y,
    const typename optox::DTensor<T, 2>::ConstRef x)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < x.size_[0] && iy < x.size_[1])
    {

        const int xp = ix + (ix < x.size_[0] - 1);
        const int yp = iy + (iy < x.size_[1] - 1);

        y(ix, iy, 0) = x(xp, iy) - x(ix, iy);
        y(ix, iy, 1) = x(ix, yp) - x(ix, iy);
    }
}

template<typename T>
__global__ void forward_differences(
    typename optox::DTensor<T, 4>::Ref y,
    const typename optox::DTensor<T, 3>::ConstRef x)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int iz = blockDim.z * blockIdx.z + threadIdx.z;
  
    if (ix < x.size_[0] && iy < x.size_[1] && iz < x.size_[2])
    {
        const int xp = ix + (ix < x.size_[0] - 1);
        const int yp = iy + (iy < x.size_[1] - 1);
        const int zp = iz + (iz < x.size_[2] - 1);

        y(ix, iy, iz, 0) = x(xp, iy, iz) - x(ix, iy, iz);
        y(ix, iy, iz, 1) = x(ix, yp, iz) - x(ix, iy, iz);
        y(ix, iy, iz, 2) = x(ix, iy, zp) - x(ix, iy, iz);
    }
}

template<typename T, unsigned int N>
void optox::NablaOperator<T, N>::computeForward(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto x = this->template getInput<T, N>(0, inputs);
    auto y = this->template getOutput<T, N+1>(0, outputs);

    if (y->size()[N] != N)
        THROW_OPTOXEXCEPTION("NablaOperator: unsupported size");

    dim3 dim_block;
    if (N == 2)
        dim_block = dim3(32, 32);
    else if (N == 3)
        dim_block = dim3(16, 16, 3);
    else
        THROW_OPTOXEXCEPTION("NablaOperator: unsupported dimension");

    dim3 dim_grid(divUp(x->size()[0], dim_block.x),
                  divUp(x->size()[1], dim_block.y),
                  divUp(x->size()[2], dim_block.z));

    forward_differences<T> <<<dim_grid, dim_block, 0, this->stream_>>>(*y, *x);
    OPTOX_CUDA_CHECK;
}


template<typename T>
__global__ void backward_differences(
    typename optox::DTensor<T, 2>::Ref x,
    const typename optox::DTensor<T, 3>::ConstRef y)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < x.size_[0] && iy < x.size_[1])
    {
        T div = (ix > 0) ? 
                        (ix < x.size_[0] - 1) ?
                                            -y(ix, iy, 0) + y(ix - 1, iy, 0)
                                            :
                                            y(ix - 1, iy, 0)
                        :
                        -y(ix, iy, 0);

        div += (iy > 0) ? 
                        (iy < x.size_[1] - 1) ?
                                            -y(ix, iy, 1) + y(ix, iy - 1, 1)
                                            :
                                            y(ix, iy - 1, 1)
                        :
                        -y(ix, iy, 1);

        x(ix, iy) = div;
    }
}

template<typename T>
__global__ void backward_differences(
    typename optox::DTensor<T, 3>::Ref x,
    const typename optox::DTensor<T, 4>::ConstRef y)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int iz = blockDim.z * blockIdx.z + threadIdx.z;
  
    if (ix < x.size_[0] && iy < x.size_[1] && iz < x.size_[2])
    {
        T div = (ix > 0) ? 
                        (ix < x.size_[0] - 1) ?
                                            -y(ix, iy, iz, 0) + y(ix - 1, iy, iz, 0)
                                            :
                                            y(ix - 1, iy, iz, 0)
                        :
                        -y(ix, iy, iz, 0);

        div += (iy > 0) ? 
                        (iy < x.size_[1] - 1) ?
                                            -y(ix, iy, iz, 1) + y(ix, iy - 1, iz, 1)
                                            :
                                            y(ix, iy - 1, iz, 1)
                        :
                        -y(ix, iy, iz, 1);

        div += (iz > 0) ? 
                        (iz < x.size_[2] - 1) ?
                                            -y(ix, iy, iz, 2) + y(ix, iy, iz - 1, 2)
                                            :
                                            y(ix, iy, iz - 1, 2)
                        :
                        -y(ix, iy, iz, 2);

        x(ix, iy, iz) = div;
    }
}

template<typename T, unsigned int N>
void optox::NablaOperator<T, N>::computeAdjoint(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto y = this->template getInput<T, N+1>(0, inputs);
    auto x = this->template getOutput<T, N>(0, outputs);

    if (y->size()[N] != N)
        THROW_OPTOXEXCEPTION("NablaOperator: unsupported size");

    dim3 dim_block;
    if (N == 2)
        dim_block = dim3(32, 32);
    else if (N == 3)
        dim_block = dim3(16, 16, 3);
    else
        THROW_OPTOXEXCEPTION("NablaOperator: unsupported dimension");

    dim3 dim_grid(divUp(x->size()[0], dim_block.x),
                  divUp(x->size()[1], dim_block.y),
                  divUp(x->size()[2], dim_block.z));

    backward_differences<T> <<<dim_grid, dim_block, 0, this->stream_>>>(*x, *y);
    OPTOX_CUDA_CHECK;
}


#define REGISTER_OP_T(T, N) \
    template class optox::NablaOperator<T, N>;;

#define REGISTER_OP(T) \
    REGISTER_OP_T(T, 2) \
    REGISTER_OP_T(T, 3)

OPTOX_CALL_REAL_NUMBER_TYPES(REGISTER_OP);
#undef REGISTER_OP
#undef REGISTER_OP_T
