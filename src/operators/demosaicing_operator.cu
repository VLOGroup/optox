#include "hip/hip_runtime.h"
///@file demosaicing_operator.cu
///@brief demosaicing operator
///@author Joana Grah <joana.grah@icg.tugraz.at>
///@date 09.07.2018


#include <iu/iucore.h>
#include <iu/iumath.h>

#include "demosaicing_operator.h"

template<typename T, optox::BayerPattern P>
__global__ void demosaicingForwardKernel(
    typename iu::LinearDeviceMemory<T, 4>::KernelData output,
    const typename iu::LinearDeviceMemory<T, 4>::KernelData input)
{
    const int x = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
    const int y = 2 * (threadIdx.y + blockIdx.y * blockDim.y);
    const int s = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x < input.size_[2] && y < input.size_[1] && s < input.size_[0])
    {
        switch (P)
        {
            case optox::BayerPattern::BGGR:
            {
                output(s, y, x, 0) = input(s, y, x, 2);
                output(s, y, x+1, 0) = input(s, y, x+1, 1);
                output(s, y+1, x, 0) = input(s, y+1, x, 1);
                output(s, y+1, x+1, 0) = input(s, y+1, x+1, 0);
                break;
            }
            case optox::BayerPattern::RGGB:
            {
                output(s, y, x, 0) = input(s, y, x, 0);
                output(s, y, x+1, 0) = input(s, y, x+1, 1);
                output(s, y+1, x, 0) = input(s, y+1, x, 1);
                output(s, y+1, x+1, 0) = input(s, y+1, x+1, 2);
                break;
            }
            case optox::BayerPattern::GBRG:
            {
                output(s, y, x, 0) = input(s, y, x, 1);
                output(s, y, x+1, 0) = input(s, y, x+1, 2);
                output(s, y+1, x, 0) = input(s, y+1, x, 0);
                output(s, y+1, x+1, 0) = input(s, y+1, x+1, 1);
                break;
            }
            case optox::BayerPattern::GRBG:
            {
                output(s, y, x, 0) = input(s, y, x, 1);
                output(s, y, x+1, 0) = input(s, y, x+1, 0);
                output(s, y+1, x, 0) = input(s, y+1, x, 2);
                output(s, y+1, x+1, 0) = input(s, y+1, x+1, 1);
                break;
            }
        }
    }
}

template<typename T>
void optox::DemosaicingOperator<T>::computeForward(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 4>(0, inputs);
    auto output = this->template getOutput<T, 4>(0, outputs);

    if (input->size()[3] != 3)
        THROW_IUEXCEPTION("DemosaicingOperator: input to forward must be RGB image!");

    if (output->size()[3] != 1)
        THROW_IUEXCEPTION("DemosaicingOperator: output of forward must have 1 channel!");

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid(iu::divUp(input->size()[2] / 2, dim_block.x),
                  iu::divUp(input->size()[1] / 2, dim_block.y),
                  iu::divUp(input->size()[0], dim_block.z));

    switch (this->pattern_)
    {
        case optox::BayerPattern::BGGR:
            demosaicingForwardKernel<T, optox::BayerPattern::BGGR> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::RGGB:
            demosaicingForwardKernel<T, optox::BayerPattern::RGGB> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GBRG:
            demosaicingForwardKernel<T, optox::BayerPattern::GBRG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GRBG:
            demosaicingForwardKernel<T, optox::BayerPattern::GRBG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
    }
    IU_CUDA_CHECK;
}

template<typename T, optox::BayerPattern P>
__global__ void demosaicingAdjointKernel(
    typename iu::LinearDeviceMemory<T, 4>::KernelData output,
    const typename iu::LinearDeviceMemory<T, 4>::KernelData input)
{
    const int x = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
    const int y = 2 * (threadIdx.y + blockIdx.y * blockDim.y);
    const int s = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x < input.size_[2] && y < input.size_[1] && s < input.size_[0])
    {
        switch (P)
        {
            case optox::BayerPattern::BGGR:
            {
                output(s, y, x, 2) = input(s, y, x, 0);
                output(s, y, x+1, 1) = input(s, y, x+1, 0);
                output(s, y+1, x, 1) = input(s, y+1, x, 0);
                output(s, y+1, x+1, 0) = input(s, y+1, x+1, 0);
                break;
            }
            case optox::BayerPattern::RGGB:
            {
                output(s, y, x, 0) = input(s, y, x, 0);
                output(s, y, x+1, 1) = input(s, y, x+1, 0);
                output(s, y+1, x, 1) = input(s, y+1, x, 0);
                output(s, y+1, x+1, 2) = input(s, y+1, x+1, 0);
                break;
            }
            case optox::BayerPattern::GBRG:
            {
                output(s, y, x, 1) = input(s, y, x, 0);
                output(s, y, x+1, 2) = input(s, y, x+1, 0);
                output(s, y+1, x, 0) = input(s, y+1, x, 0);
                output(s, y+1, x+1, 1) = input(s, y+1, x+1, 0);
                break;
            }
            case optox::BayerPattern::GRBG:
            {
                output(s, y, x, 1) = input(s, y, x, 0);
                output(s, y, x+1, 0) = input(s, y, x+1, 0);
                output(s, y+1, x, 2) = input(s, y+1, x, 0);
                output(s, y+1, x+1, 1) = input(s, y+1, x+1, 0);
                break;
            }
        }
    }
}

template<typename T>
void optox::DemosaicingOperator<T>::computeAdjoint(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 4>(0, inputs);
    auto output = this->template getOutput<T, 4>(0, outputs);

    if (input->size()[3] != 1)
        THROW_IUEXCEPTION("DemosaicingOperator: input to adjoint must have 1 channel!");

    if (output->size()[3] != 3)
        THROW_IUEXCEPTION("DemosaicingOperator: output of adjoint must be RGB image!");

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid(iu::divUp(input->size()[2] / 2, dim_block.x),
                  iu::divUp(input->size()[1] / 2, dim_block.y),
                  iu::divUp(input->size()[0], dim_block.z));

    switch (this->pattern_)
    {
        case optox::BayerPattern::BGGR:
            demosaicingAdjointKernel<T, optox::BayerPattern::BGGR> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::RGGB:
            demosaicingAdjointKernel<T, optox::BayerPattern::RGGB> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GBRG:
            demosaicingAdjointKernel<T, optox::BayerPattern::GBRG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GRBG:
            demosaicingAdjointKernel<T, optox::BayerPattern::GRBG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
    }
    IU_CUDA_CHECK;
}

#define REGISTER_OP(T) \
    template class optox::DemosaicingOperator<T>;

OPTOX_CALL_REAL_NUMBER_TYPES(REGISTER_OP);
#undef REGISTER_OP
#undef REGISTER_OP_T
