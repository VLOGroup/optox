#include "hip/hip_runtime.h"
///@file demosaicing_operator.cu
///@brief demosaicing operator
///@author Joana Grah <joana.grah@icg.tugraz.at>
///@date 09.07.2018


#include <iu/iucore.h>
#include <iu/iumath.h>

#include "demosaicing_operator.h"

template<typename T, optox::BayerPattern P>
__global__ void demosaicingForwardKernel(
    typename iu::LinearDeviceMemory<T, 4>::KernelData output,
    const typename iu::LinearDeviceMemory<T, 4>::KernelData input)
{
    const int x = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
    const int y = 2 * (threadIdx.y + blockIdx.y * blockDim.y);
    const int s = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x < input.size_[1] && y < input.size_[2] && s < input.size_[3])
    {
        switch (P)
        {
            case optox::BayerPattern::BGGR:
            {
                output(0, x, y, s) = input(2, x, y, s);
                output(0, x+1, y, s) = input(1, x+1, y, s);
                output(0, x, y+1, s) = input(1, x, y+1, s);
                output(0, x+1, y+1, s) = input(0, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::RGGB:
            {
                output(0, x, y, s) = input(0, x, y, s);
                output(0, x+1, y, s) = input(1, x+1, y, s);
                output(0, x, y+1, s) = input(1, x, y+1, s);
                output(0, x+1, y+1, s) = input(2, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::GBRG:
            {
                output(0, x, y, s) = input(1, x, y, s);
                output(0, x+1, y, s) = input(2, x+1, y, s);
                output(0, x, y+1, s) = input(0, x, y+1, s);
                output(0, x+1, y+1, s) = input(1, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::GRBG:
            {
                output(0, x, y, s) = input(1, x, y, s);
                output(0, x+1, y, s) = input(0, x+1, y, s);
                output(0, x, y+1, s) = input(2, x, y+1, s);
                output(0, x+1, y+1, s) = input(1, x+1, y+1, s);
                break;
            }
        }
    }
}

template<typename T>
void optox::DemosaicingOperator<T>::computeForward(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 4>(0, inputs);
    auto output = this->template getOutput<T, 4>(0, outputs);

    if (input->size()[0] != 3)
        THROW_IUEXCEPTION("DemosaicingOperator: input to forward must be RGB image!");

    if (output->size()[0] != 1)
        THROW_IUEXCEPTION("DemosaicingOperator: output of forward must have 1 channel!");

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid(iu::divUp(input->size()[1] / 2 + 1, dim_block.x),
                  iu::divUp(input->size()[2] / 2 + 1, dim_block.y),
                  iu::divUp(input->size()[3], dim_block.z));

    switch (this->pattern_)
    {
        case optox::BayerPattern::BGGR:
            demosaicingForwardKernel<T, optox::BayerPattern::BGGR> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::RGGB:
            demosaicingForwardKernel<T, optox::BayerPattern::RGGB> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GBRG:
            demosaicingForwardKernel<T, optox::BayerPattern::GBRG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GRBG:
            demosaicingForwardKernel<T, optox::BayerPattern::GRBG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
    }
    IU_CUDA_CHECK;
}

template<typename T, optox::BayerPattern P>
__global__ void demosaicingAdjointKernel(
    typename iu::LinearDeviceMemory<T, 4>::KernelData output,
    const typename iu::LinearDeviceMemory<T, 4>::KernelData input)
{
    const int x = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
    const int y = 2 * (threadIdx.y + blockIdx.y * blockDim.y);
    const int s = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (x < input.size_[1] && y < input.size_[2] && s < input.size_[3])
    {
        switch (P)
        {
            case optox::BayerPattern::BGGR:
            {
		output(2, x, y, s) = input(0, x, y, s);
                output(1, x+1, y, s) = input(0, x+1, y, s);
                output(1, x, y+1, s) = input(0, x, y+1, s);
                output(0, x+1, y+1, s) = input(0, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::RGGB:
            {
		output(0, x, y, s) = input(0, x, y, s);
                output(1, x+1, y, s) = input(0, x+1, y, s);
                output(1, x, y+1, s) = input(0, x, y+1, s);
                output(2, x+1, y+1, s) = input(0, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::GBRG:
            {	
		output(1, x, y, s) = input(0, x, y, s);
                output(2, x+1, y, s) = input(0, x+1, y, s);
                output(0, x, y+1, s) = input(0, x, y+1, s);
                output(1, x+1, y+1, s) = input(0, x+1, y+1, s);
                break;
            }
            case optox::BayerPattern::GRBG:
            {
		output(1, x, y, s) = input(0, x, y, s);
                output(0, x+1, y, s) = input(0, x+1, y, s);
                output(2, x, y+1, s) = input(0, x, y+1, s);
                output(1, x+1, y+1, s) = input(0, x+1, y+1, s);
                break;
            }
        }
    }
}

template<typename T>
void optox::DemosaicingOperator<T>::computeAdjoint(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 4>(0, inputs);
    auto output = this->template getOutput<T, 4>(0, outputs);

    if (input->size()[0] != 1)
        THROW_IUEXCEPTION("DemosaicingOperator: input to adjoint must have 1 channel!");

    if (output->size()[0] != 3)
        THROW_IUEXCEPTION("DemosaicingOperator: output of adjoint must be RGB image!");

    iu::math::fill(*output, static_cast<T>(0));

    dim3 dim_block = dim3(32, 32, 1);
    dim3 dim_grid(iu::divUp(input->size()[1] / 2 + 1, dim_block.x),
                  iu::divUp(input->size()[2] / 2 + 1, dim_block.y),
                  iu::divUp(input->size()[3], dim_block.z));

    switch (this->pattern_)
    {
        case optox::BayerPattern::BGGR:
            demosaicingAdjointKernel<T, optox::BayerPattern::BGGR> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::RGGB:
            demosaicingAdjointKernel<T, optox::BayerPattern::RGGB> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GBRG:
            demosaicingAdjointKernel<T, optox::BayerPattern::GBRG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
        case optox::BayerPattern::GRBG:
            demosaicingAdjointKernel<T, optox::BayerPattern::GRBG> <<<dim_grid, dim_block>>>(*output, *input);
            break;
    }
    IU_CUDA_CHECK;
}

#define REGISTER_OP(T) \
    template class optox::DemosaicingOperator<T>;

OPTOX_CALL_REAL_NUMBER_TYPES(REGISTER_OP);
#undef REGISTER_OP
#undef REGISTER_OP_T