#include "hip/hip_runtime.h"
///@file act_rbf.cu
///@brief Gaussian radial basis function operator
///@author Erich Kobler <erich.kobler@icg.tugraz.at>
///@date 01.2019


#include <iu/iucore.h>
#include <iu/iumath.h>
#include <type_traits>

#include "act_rbf.h"
#include "utils.cuh"


// forward Gaussian rbf
template<typename T>
__global__ void act_rbf_forward_kernel(
    typename iu::LinearDeviceMemory<T, 2>::KernelData output,
    const typename iu::LinearDeviceMemory<T, 2>::KernelData input,
    const typename iu::LinearDeviceMemory<T, 2>::KernelData weights,
    T vmin, T vmax)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= input.size_[0] || y >= input.size_[1])
        return;

    const int Nw = weights.size_[0];

    const T sigma = (vmax - vmin) / (Nw - 1);
    const T sigma2 = sigma * sigma;
    const T k = ((vmax - vmin) / (Nw - 1));

    T inp_pos = input(x, y);
    T val = 0;
    for (int i = 0; i < Nw; ++i)
    {
        // compute the base function
        const T mu = k * i + vmin;
        T base_function = 0;
        const T diff = inp_pos - mu;
        if (std::is_same<T, float>::value)
            base_function = expf( -(diff*diff) / (sigma2 * 2)) * 0.4;
        else
            base_function = exp( -(diff*diff) / (sigma2 * 2)) * 0.4;
        val += weights(i, y) * base_function;
    }

    output(x, y) = val;
}


// backward Gaussian rbf
template<typename T>
__global__ void act_rbf_backward_kernel(
    typename iu::LinearDeviceMemory<T, 2>::KernelData grad_input,
    typename iu::LinearDeviceMemory<T, 2>::KernelData grad_weights,
    const typename iu::LinearDeviceMemory<T, 2>::KernelData input,
    const typename iu::LinearDeviceMemory<T, 2>::KernelData weights,
    const typename iu::LinearDeviceMemory<T, 2>::KernelData grad_output,
    T vmin, T vmax)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    extern __shared__ __align__(sizeof(T)) unsigned char sbuffer[];
    T *sdata = reinterpret_cast<T*>(sbuffer);

    if (x >= input.size_[0] || y >= input.size_[1])
    {
        sdata[tid] = 0;
        return;
    }

    const int Nw = weights.size_[0];

    const T sigma = (vmax - vmin) / (Nw - 1);
    const T sigma2 = sigma * sigma;
    const T k = ((vmax - vmin) / (Nw - 1));

    T inp_pos = input(x, y);
    T grad_out_pos = grad_output(x, y);
    T grad_inp = 0;
    for (int i = 0; i < Nw; ++i)
    {
        // compute the base function and its derivative
        const T mu = k * i + vmin;
        T base_function = 0;
        T base_function_prime = 0;
        const T diff = inp_pos - mu;
        if (std::is_same<T, float>::value)
            base_function = expf( -(diff*diff) / (sigma2 * 2)) * 0.4;
        else
            base_function = exp( -(diff*diff) / (sigma2 * 2)) * 0.4;
        base_function_prime = base_function * (-diff)/sigma2;
        // backpropagate the gradient to the input
        grad_inp += weights(i, y) * base_function_prime;

        // backpropagate the gradient to a single weight
        sdata[tid] = base_function * grad_out_pos;

        // parallel reduction along outer dimensions
        parallelReduce(sdata, tid, blockDim.x);

        if(tid == 0)
            atomicAdd(&(grad_weights(i, y)), sdata[tid]);
    }
    grad_input(x, y) = grad_inp;
}


template<typename T>
void optox::RBFActOperator<T>::computeForward(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 2>(0, inputs);
    auto weights = this->template getInput<T, 2>(1, inputs);

    auto output = this->template getOutput<T, 2>(0, outputs);

    this->checkSize(input->size(), weights->size());

    int thread_per_block = 256;
    dim3 dim_block = dim3(thread_per_block, 1);
    dim3 block_count = dim3(iu::divUp(input->size()[0], dim_block.x),
                            input->size()[1]);

    std::cout << "input " << *input << std::endl;
    std::cout << "weights " << *weights << std::endl;
    act_rbf_forward_kernel<T><<<dim_block, block_count, 0, this->stream_>>>(
        *output,
        *input, *weights,
        this->vmin_, this->vmax_);
}

template<typename T>
void optox::RBFActOperator<T>::computeAdjoint(optox::OperatorOutputVector &&outputs,
    const optox::OperatorInputVector &inputs)
{
    auto input = this->template getInput<T, 2>(0, inputs);
    auto weights = this->template getInput<T, 2>(1, inputs);
    auto grad_output = this->template getInput<T, 2>(2, inputs);

    auto grad_input = this->template getOutput<T, 2>(0, outputs);
    auto grad_weights = this->template getOutput<T, 2>(1, outputs);

    this->checkSize(input->size(), weights->size());

    int thread_per_block = 256;
    dim3 dim_block = dim3(thread_per_block, 1);
    dim3 block_count = dim3(iu::divUp(input->size()[0], dim_block.x),
                            input->size()[1]);

    act_rbf_backward_kernel<T><<<dim_block, block_count, thread_per_block * sizeof(T), this->stream_>>>(
        *grad_input, *grad_weights,
        *input, *weights, *grad_output,
        this->vmin_, this->vmax_);
}


#define REGISTER_OP(T) \
    template class optox::RBFActOperator<T>;

OPTOX_CALL_REAL_NUMBER_TYPES(REGISTER_OP);
#undef REGISTER_OP
